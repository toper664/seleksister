
#include <hip/hip_runtime.h>
__global__ void grayscale(const float* image_in, float* image_out, int rows, int cols) {

    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    if (row < rows && col < cols) {

        int i = (row * cols + col) * 3;
        
        float gray = static_cast<float>(image_in[i] + image_in[i + 1] + image_in[i + 2]) / 3.0;

        image_out[i] = gray;
        image_out[i + 1] = gray;
        image_out[i + 2] = gray;

    }

}

__global__ void contrast(const float* image_in, float* image_out, int rows, int cols, int c) {

    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int rgb = threadIdx.z;

    if (row < rows && col < cols) {

        int i = (row * cols + col) * 3 + rgb;
        
        float f = static_cast<float>(259 * (c + 255)) / static_cast<float>(255 * (259 - c));

        float result = f * (image_in[i] - 128) + 128;
        result = (result > 255) ? 255 : result;
        result = (result < 0) ? 0 : result;

        image_out[i] = result;

    }

}

__global__ void saturation(const float* image_in, float* image_out, int rows, int cols, float s) {

    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    if (row < rows && col < cols) {

        int i = (row * cols + col);
        
        image_out[i] = image_in[i] * s;

    }

}

// makasih bang aw hehehe
__global__ void blur(const float* image_in, float* image_out, int rows, int cols, int v) {

    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int rgb = threadIdx.z;

    int lim = v * 3;

    if (row < rows && col < cols) {

        int idx = (row * cols + col) * 3 + rgb;

        int sum = 0;

        if (col < lim || col >= ((cols * 3) - lim)) {
            for (int i = -v; i <= v; i++) {
                for (int j = -lim; j <= lim; j+= 3) {
                    if (row + i >= 0) {
                        sum += image_in[idx + (i * (cols * 3) - j)];
                    }
                }
            }
            image_out[idx] = sum / (1 + v*2) / (1 + v*2);
        } else if (row < v || row >= (rows - v)) {
            for (int i = -v; i <= v; i++) {
                for (int j = -lim; j <= lim; j+= 3) {
                    if (col + j >= 0) {
                        sum += image_in[idx + ((-1)*i * (cols * 3) - j)];
                    }
                }
            }
            image_out[idx] = sum / (1 + v*2) / (1 + v*2);
        } else {
            for (int i = -v; i <= v; i++) {
                for (int j = -lim; j <= lim; j+= 3) {
                    sum += image_in[idx + (i * (cols * 3) + j)];
                }
            }
            image_out[idx] = sum / (1 + v*2) / (1 + v*2);
        }

    }

}

// makasih bang AW hehehe
__constant__ float sobel_x[9] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
__constant__ float sobel_y[9] = {1, 2, 1, 0, 0, 0, -1, -2, -1};

__global__ void edge_detection(const float* image_in, float* image_out, int rows, int cols) {

    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    float dx, dy;
    if (row < rows && col < cols) {

        int idx = (row * cols + col) * 3; 
        
        if ( !(col < 3 || col >= ((cols * 3) - 3) || row < 1 || row >= (rows - 1)) ){
            int previdx = idx - (cols * 3);
            int nextidx = idx + (cols * 3);
            
            float val0 = image_in[previdx - 3] + image_in[previdx - 2] + image_in[previdx - 1];
            float val1 = image_in[previdx] + image_in[idx + 1] + image_in[previdx + 2];
            float val2 = image_in[previdx + 3] + image_in[previdx + 4] + image_in[previdx + 5];

            float val3 = image_in[idx - 3] + image_in[idx - 2] + image_in[idx - 1];

            float val5 = image_in[idx + 3] + image_in[idx + 4] + image_in[idx + 5];
            
            float val6 = image_in[nextidx - 3] + image_in[nextidx - 2] + image_in[nextidx - 1];
            float val7 = image_in[nextidx] + image_in[nextidx + 1] + image_in[nextidx + 2];
            float val8 = image_in[nextidx + 3] + image_in[nextidx + 4] + image_in[nextidx + 5];

            float xval = (sobel_x[0] * val0) + (sobel_x[2] * val2) + (sobel_x[3] * val3) + (sobel_x[5] * val5) + (sobel_x[6] * val6) + (sobel_x[8] * val8);
            float yval = (sobel_y[0] * val0) + (sobel_y[1] * val1) + (sobel_y[2] * val2) + (sobel_y[6] * val6) + (sobel_y[7] * val7) + (sobel_y[8] * val8);
            
            float magnitude = abs(xval) + abs(yval);
            
            image_out[idx] = magnitude;
            image_out[idx + 1] = magnitude;
            image_out[idx + 2] = magnitude;
        }

    }

}